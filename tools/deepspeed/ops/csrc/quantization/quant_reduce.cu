#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// SPDX-License-Identifier: Apache-2.0

// DeepSpeed Team

#include <cstdio>
#include "dequantization_utils.h"
#include "ds_kernel_utils.h"
#include "memory_access_utils.h"
#include "quantization_utils.h"
#include "reduction_utils.h"

using rop = reduce::ROpType;

/*
TODO(cmikeh2): Add implementation that better handles larger nodes. It would like make sense
to leverage some parallel reductions here to improve performance.
*/

template <int numBits, int numTensors, int totalChunks, quantize::Type quantType>
__global__ void __launch_bounds__(1024) dequant_reduce(int8_t* reduced_data,
                                                       float* reduced_scales,
                                                       const int8_t* input_data,
                                                       const float* input_scales,
                                                       int elems_per_out_group,
                                                       int elems_per_in_tensor,
                                                       int groups_per_in_tensor,
                                                       int elems_per_in_group,
                                                       int num_tensors)
{
    cg::thread_block tb = cg::this_thread_block();
    cg::thread_block_tile<hw_warp_size> warp = cg::tiled_partition<hw_warp_size>(tb);

    // NOTE(cmikeh2): This probably could be hardcoded to a larger number,
    // but that means even stronger restrictions on the number of elements per group
    // A performance analysis here might be beneficial
    constexpr int mem_granularity = (numBits == 8) ? 8 : 4;
    constexpr int elems_per_load = mem_granularity / sizeof(int8_t);  // div by 1
    constexpr int storage_values = 16 / sizeof(__half2);

    const int block_offset = tb.group_index().x * elems_per_out_group;
    const int elem_offset = tb.thread_index().x * elems_per_load;
    const int base_offset = block_offset + elem_offset;
    const int stride = tb.group_dim().x * elems_per_load;

    __half2 local_buffer[totalChunks * storage_values];

    quantize::GroupStats<quantType> stats;

#pragma unroll
    for (int i = 0; i < totalChunks; i++) {
        __half2* iteration_buffer = local_buffer + i * storage_values;

#pragma unroll
        for (int j = 0; j < storage_values; j++) {
            iteration_buffer[j] = reduce::init<rop::Add, __half2>();
        }

        const int iter_offset = i * stride + base_offset;
        const int iter_scale_idx = iter_offset / elems_per_in_group;
        bool do_loads = i * stride + elem_offset < elems_per_out_group;

        if (numTensors > 0) {
#pragma unroll
            for (int j = 0; j < numTensors; j++) {
                if (do_loads) {
                    int8_t load_buffer[elems_per_load];

                    mem_access::load_global<mem_granularity>(
                        load_buffer, input_data + j * elems_per_in_tensor + iter_offset);

                    quantize::Params<quantType, numBits> params(
                        input_scales + j * groups_per_in_tensor, iter_scale_idx);

                    __half2 dequant_buffer[storage_values];
                    dequantize::chunk<numBits, quantType>(dequant_buffer, load_buffer, params);

#pragma unroll
                    for (int k = 0; k < storage_values; k++) {
                        iteration_buffer[k] =
                            reduce::element<rop::Add>(iteration_buffer[k], dequant_buffer[k]);
                    }
                }
            }
        } else {
#pragma unroll 4
            for (int j = 0; j < num_tensors; j++) {
                if (do_loads) {
                    int8_t load_buffer[elems_per_load];

                    mem_access::load_global<mem_granularity>(
                        load_buffer, input_data + j * elems_per_in_tensor + iter_offset);

                    quantize::Params<quantType, numBits> params(
                        input_scales + j * groups_per_in_tensor, iter_scale_idx);

                    __half2 dequant_buffer[storage_values];
                    dequantize::chunk<numBits, quantType>(dequant_buffer, load_buffer, params);

#pragma unroll
                    for (int k = 0; k < storage_values; k++) {
                        iteration_buffer[k] =
                            reduce::element<rop::Add>(iteration_buffer[k], dequant_buffer[k]);
                    }
                }
            }
        }

#pragma unroll
        for (int j = 0; j < storage_values; j++) { stats.update(iteration_buffer[j]); }
    }

    auto params = stats.template get_params<numBits, 1024>(tb, warp);

    if (tb.thread_index().x == 0) { params.store(reduced_scales, tb.group_index().x); }

#pragma unroll
    for (int i = 0; i < totalChunks; i++) {
        const int iter_offset = i * stride + base_offset;
        if (i * stride + elem_offset < elems_per_out_group) {
            int8_t local_output[elems_per_load];
            quantize::_chunk<numBits, quantType>(
                local_output, local_buffer + i * storage_values, params);
            mem_access::store_global<mem_granularity>(reduced_data + iter_offset, local_output);
        }
    }
}

template <int Power>
int32_t pow2_round(int32_t raw_value)
{
    return (((raw_value - 1) >> Power) + 1) << Power;
}

#define LAUNCH_DEQUANT_REDUCE(num_chunks)                      \
    dequant_reduce<numBits, numTensors, num_chunks, quantType> \
        <<<grid, block, 0, stream>>>(reduced_data,             \
                                     reduced_scales,           \
                                     input_data,               \
                                     input_scales,             \
                                     elems_per_out_group,      \
                                     elems_per_in_tensor,      \
                                     groups_per_in_tensor,     \
                                     elems_per_in_group,       \
                                     num_tensors);

template <int numBits, int numTensors, quantize::Type quantType>
void launch_dequant_reduce_impl(int8_t* reduced_data,
                                float* reduced_scales,
                                const int8_t* input_data,
                                const float* input_scales,
                                int out_groups,
                                int elems_per_out_group,
                                int elems_per_in_tensor,
                                int groups_per_in_tensor,
                                int elems_per_in_group,
                                int num_tensors,
                                hipStream_t stream)
{
    // This is a coincidence. This is derived by 8 halves per 16 bytes with 2-way packing for int4
    constexpr int elems_per_thread = numBits;
    const int one_step_threads =
        next_pow2((elems_per_out_group + elems_per_thread - 1) / (elems_per_thread));
    // TODO(cmikeh2): Tune this
    const int threads = (one_step_threads < 1024) ? one_step_threads : 1024;

    dim3 block(threads);
    dim3 grid(out_groups);

    const int elems_per_step = threads * elems_per_thread;
    const int unroll_raw = (elems_per_out_group + elems_per_step - 1) / elems_per_step;

    const int unroll = (unroll_raw >= 4) ? pow2_round<1>(unroll_raw) : unroll_raw;

    if (unroll == 1) {
        // 0-4096 elems
        LAUNCH_DEQUANT_REDUCE(1);
    } else if (unroll == 2) {
        // 4097-8192 etc...
        LAUNCH_DEQUANT_REDUCE(2);
    } else if (unroll == 3) {
        LAUNCH_DEQUANT_REDUCE(3);
    } else if (unroll == 4) {
        LAUNCH_DEQUANT_REDUCE(4);
    } else if (unroll == 6) {
        LAUNCH_DEQUANT_REDUCE(6);
    } else if (unroll == 8) {
        LAUNCH_DEQUANT_REDUCE(8);
    } else if (unroll == 10) {
        LAUNCH_DEQUANT_REDUCE(10);
    } else if (unroll == 12) {
        // 48k limit
        LAUNCH_DEQUANT_REDUCE(12);
    } else {
        assert(false);
    }
}

#define LAUNCH_DEQUANT_REDUCE_IMPL(NUM_BITS, NUM_GPUS, QUANT_TYPE)                   \
    launch_dequant_reduce_impl<NUM_BITS, NUM_GPUS, QUANT_TYPE>(reduced_data,         \
                                                               reduced_scales,       \
                                                               input_data,           \
                                                               input_scales,         \
                                                               out_groups,           \
                                                               elems_per_out_group,  \
                                                               elems_per_in_tensor,  \
                                                               groups_per_in_tensor, \
                                                               elems_per_in_group,   \
                                                               num_gpus,             \
                                                               stream);

void launch_dequant_reduce(int8_t* reduced_data,
                           float* reduced_scales,
                           const int8_t* input_data,
                           const float* input_scales,
                           int num_gpus,
                           int num_bits,
                           quantize::Type quant_type,
                           int out_groups,
                           int elems_per_out_group,
                           int elems_per_in_tensor,
                           int groups_per_in_tensor,
                           int elems_per_in_group,
                           hipStream_t stream)
{
    if (quant_type == quantize::Type::Symmetric) {
        if (num_bits == 4) {
            if (num_gpus == 8) {
                LAUNCH_DEQUANT_REDUCE_IMPL(4, 8, quantize::Type::Symmetric);
            } else if (num_gpus == 16) {
                LAUNCH_DEQUANT_REDUCE_IMPL(4, 16, quantize::Type::Symmetric);
            } else {
                LAUNCH_DEQUANT_REDUCE_IMPL(4, -1, quantize::Type::Symmetric);
            }
        } else if (num_bits == 8) {
            if (num_gpus == 8) {
                LAUNCH_DEQUANT_REDUCE_IMPL(8, 8, quantize::Type::Symmetric);
            } else if (num_gpus == 16) {
                LAUNCH_DEQUANT_REDUCE_IMPL(8, 16, quantize::Type::Symmetric);
            } else {
                LAUNCH_DEQUANT_REDUCE_IMPL(8, -1, quantize::Type::Symmetric);
            }
        }
    } else if (quant_type == quantize::Type::Asymmetric) {
        if (num_bits == 4) {
            if (num_gpus == 8) {
                LAUNCH_DEQUANT_REDUCE_IMPL(4, 8, quantize::Type::Asymmetric);
            } else if (num_gpus == 16) {
                LAUNCH_DEQUANT_REDUCE_IMPL(4, 16, quantize::Type::Asymmetric);
            } else {
                LAUNCH_DEQUANT_REDUCE_IMPL(4, -1, quantize::Type::Asymmetric);
            }
        } else if (num_bits == 8) {
            if (num_gpus == 8) {
                LAUNCH_DEQUANT_REDUCE_IMPL(8, 8, quantize::Type::Asymmetric);
            } else if (num_gpus == 16) {
                LAUNCH_DEQUANT_REDUCE_IMPL(8, 16, quantize::Type::Asymmetric);
            } else {
                LAUNCH_DEQUANT_REDUCE_IMPL(8, -1, quantize::Type::Asymmetric);
            }
        }
    }
}
