#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/

// This file may have been modified by Bytedance Ltd. and/or its affiliates (“Bytedance's Modifications”). 
// All Bytedance's Modifications are Copyright 2024 Bytedance Ltd. and/or its affiliates. 

// #pragma once

#include <c10/util/BFloat16.h>
#include <c10/util/Half.h>
#include <c10/cuda/CUDAException.h>  // For C10_CUDA_CHECK and C10_CUDA_KERNEL_LAUNCH_CHECK

#include "fast_hadamard_transform.h"
#include "fast_hadamard_transform_common.h"
#include "fast_hadamard_transform_special.h"
#include "static_switch.h"

template<int kNThreads_, int kLogN_, typename input_t_>
struct fast_hadamard_transform_kernel_traits {
    using input_t = input_t_;
    static constexpr int kNThreads = kNThreads_;
    static constexpr int kLogN = kLogN_;
    static constexpr int N = 1 << kLogN;
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
    static constexpr int kNElts = kNBytes == 4 ? 4 : 8;
    // It's possible that we need to do 2 rounds of exchange if input_t is 16 bits
    // (since then we'd have 8 values of float, and each round we can exchange 4 floats).
    static constexpr int kNExchangePerVec = sizeof(float) / sizeof(input_t);
    using vec_t = typename BytesToType<kNBytes * kNElts>::Type;
    static constexpr int kNChunks = N / (kNElts * kNThreads);
    // We don't want to use more than 32 KB of shared memory.
    static constexpr int kSmemExchangeSize = std::min(N * 4, 32 * 1024);
    static constexpr int kNExchangeRounds = N * 4 / kSmemExchangeSize;
    static_assert(kNExchangeRounds * kSmemExchangeSize == N * 4);
    static constexpr int kSmemSize = kSmemExchangeSize;
};

template <int kNChunks>
__device__ __forceinline__ void hadamard_mult_thread_chunk_12(float x[kNChunks][12]) {
    #pragma unroll
    for (int c = 0; c < kNChunks; ++c) { hadamard_mult_thread_12(x[c]); }
}

template <int kNChunks>
__device__ __forceinline__ void hadamard_mult_thread_chunk_20(float x[kNChunks][20]) {
    #pragma unroll
    for (int c = 0; c < kNChunks; ++c) { hadamard_mult_thread_20(x[c]); }
}

template <int kNChunks>
__device__ __forceinline__ void hadamard_mult_thread_chunk_28(float x[kNChunks][28]) {
    #pragma unroll
    for (int c = 0; c < kNChunks; ++c) { hadamard_mult_thread_28(x[c]); }
}

template <int kNChunks>
__device__ __forceinline__ void hadamard_mult_thread_chunk_40(float x[kNChunks][40]) {
    #pragma unroll
    for (int c = 0; c < kNChunks; ++c) { hadamard_mult_thread_40(x[c]); }
}

template<typename Ktraits>
__global__ __launch_bounds__(Ktraits::kNThreads)
void fast_hadamard_transform_kernel(HadamardParamsBase params) {
    constexpr int kNThreads = Ktraits::kNThreads;
    constexpr int kNElts = Ktraits::kNElts;
    constexpr int kNExchangePerVec = Ktraits::kNExchangePerVec;
    constexpr int kNExchangeRounds = Ktraits::kNExchangeRounds;
    constexpr int kNChunks = Ktraits::kNChunks;
    using input_t = typename Ktraits::input_t;
    using vec_t = typename Ktraits::vec_t;

    constexpr int kLogNElts = cilog2(Ktraits::kNElts);
    static_assert(1 << kLogNElts == kNElts, "kNElts must be a power of 2");
    constexpr int kWarpSize = std::min(kNThreads, 32);
    constexpr int kLogWarpSize = cilog2(kWarpSize);
    static_assert(1 << kLogWarpSize == kWarpSize, "Warp size must be a power of 2");
    constexpr int kNWarps = kNThreads / kWarpSize;
    constexpr int kLogNWarps = cilog2(kNWarps);
    static_assert(1 << kLogNWarps == kNWarps, "kNWarps must be a power of 2");
    constexpr int kLoadsPerExchange = Ktraits::kSmemExchangeSize / (sizeof(vec_t) * kNThreads);
    static_assert(kLoadsPerExchange * sizeof(vec_t) * kNThreads == Ktraits::kSmemExchangeSize, "kSmemExchangeSize should be a power of 2");
    static_assert(kNExchangeRounds * kLoadsPerExchange * sizeof(vec_t) == kNChunks * kNElts * sizeof(float));

    constexpr int kChunksPerExchange = Ktraits::kSmemExchangeSize / (sizeof(vec_t) * kNExchangePerVec * kNThreads);
    static_assert(kChunksPerExchange * sizeof(vec_t) * kNExchangePerVec * kNThreads == Ktraits::kSmemExchangeSize);
    constexpr int kNExchanges = kNChunks / kChunksPerExchange;
    static_assert(kNExchanges * kChunksPerExchange == kNChunks);

    // Shared memory.
    extern __shared__ char smem_[];
    vec_t *smem_exchange = reinterpret_cast<vec_t *>(smem_);

    const int batch_id = blockIdx.x;
    input_t *x = reinterpret_cast<input_t *>(params.x_ptr) + batch_id * params.x_batch_stride;
    input_t *out = reinterpret_cast<input_t *>(params.out_ptr) + batch_id * params.out_batch_stride;

    float x_vals[kNChunks][kNElts];
    load_input<kNChunks, kNElts, input_t>(x, x_vals, params.dim);

    hadamard_mult_thread<kLogNElts, kNChunks>(x_vals);
    hadamard_mult_warp<kLogWarpSize, 0, kNChunks, kNElts>(x_vals);

    if constexpr (kNWarps > 1) {
        exchange_smem_pre<kNChunks, kChunksPerExchange, kNElts, kWarpSize, kNWarps, true, vec_t>(x_vals, smem_exchange);
        hadamard_mult_warp<kLogNWarps, 0, kNChunks, kNElts>(x_vals);
        exchange_smem_pre<kNChunks, kChunksPerExchange, kNElts, kWarpSize, kNWarps, false, vec_t>(x_vals, smem_exchange);
    }

    if constexpr (kNChunks > 1) {
        float x_vals_transposed[kNElts][kNChunks];
        #pragma unroll
        for (int c = 0; c < kNChunks; ++c) {
            #pragma unroll
            for (int i = 0; i < kNElts; ++i) { x_vals_transposed[i][c] = x_vals[c][i]; }
        }
        if constexpr (kNChunks == 12) {
            hadamard_mult_thread_chunk_12<kNElts>(x_vals_transposed);
        } else if constexpr (kNChunks == 20) {
            hadamard_mult_thread_chunk_20<kNElts>(x_vals_transposed);
        } else if constexpr (kNChunks == 28) {
            hadamard_mult_thread_chunk_28<kNElts>(x_vals_transposed);
        } else if constexpr (kNChunks == 40) {
            hadamard_mult_thread_chunk_40<kNElts>(x_vals_transposed);
        } else {
            constexpr int kLogNChunks = cilog2(kNChunks);
            static_assert(1 << kLogNChunks == kNChunks, "kNChunks must be a power of 2");
            hadamard_mult_thread<kLogNChunks, kNElts>(x_vals_transposed);
        }
        #pragma unroll
        for (int c = 0; c < kNChunks; ++c) {
            #pragma unroll
            for (int i = 0; i < kNElts; ++i) { x_vals[c][i] = x_vals_transposed[i][c]; }
        }
    }

    store_output<kNChunks, kNElts, input_t>(out, x_vals, params.dim, params.scale);
}

template<int kNThreads, int kLogN, typename input_t>
void fast_hadamard_transform_launch(HadamardParamsBase &params, hipStream_t stream) {
    using Ktraits = fast_hadamard_transform_kernel_traits<kNThreads, kLogN, input_t>;
    constexpr int kSmemSize = Ktraits::kSmemSize;
    dim3 grid(params.batch);
    auto kernel = &fast_hadamard_transform_kernel<Ktraits>;
    if (kSmemSize >= 48 * 1024) {
        C10_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
            kernel), hipFuncAttributeMaxDynamicSharedMemorySize, kSmemSize));
        }
    kernel<<<grid, Ktraits::kNThreads, kSmemSize, stream>>>(params);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template<typename input_t>
void fast_hadamard_transform_cuda(HadamardParamsBase &params, hipStream_t stream) {
    if (params.log_N == 3) {
        fast_hadamard_transform_launch<1, 3, input_t>(params, stream);
    } else if (params.log_N == 4) {
        fast_hadamard_transform_launch<2, 4, input_t>(params, stream);
    } else if (params.log_N == 5) {
        fast_hadamard_transform_launch<4, 5, input_t>(params, stream);
    } else if (params.log_N == 6) {
        fast_hadamard_transform_launch<8, 6, input_t>(params, stream);
    } else if (params.log_N == 7) {
        fast_hadamard_transform_launch<16, 7, input_t>(params, stream);
    } else if (params.log_N == 8) {
        fast_hadamard_transform_launch<32, 8, input_t>(params, stream);
    } else if (params.log_N == 9) {
        fast_hadamard_transform_launch<32, 9, input_t>(params, stream);
    } else if (params.log_N == 10) {
        fast_hadamard_transform_launch<128, 10, input_t>(params, stream);
    } else if (params.log_N == 11) {
        fast_hadamard_transform_launch<256, 11, input_t>(params, stream);
    } else if (params.log_N == 12) {
        fast_hadamard_transform_launch<256, 12, input_t>(params, stream);
    } else if (params.log_N == 13) {
        fast_hadamard_transform_launch<256, 13, input_t>(params, stream);
    } else if (params.log_N == 14) {
        fast_hadamard_transform_launch<256, 14, input_t>(params, stream);
    } else if (params.log_N == 15) {
        fast_hadamard_transform_launch<256, 15, input_t>(params, stream);
    }
}

template void fast_hadamard_transform_cuda<float>(HadamardParamsBase &params, hipStream_t stream);
template void fast_hadamard_transform_cuda<at::Half>(HadamardParamsBase &params, hipStream_t stream);
template void fast_hadamard_transform_cuda<at::BFloat16>(HadamardParamsBase &params, hipStream_t stream);